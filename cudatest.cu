#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda/sha256.cuh"
#include "cuda/sha256.cu"
#include <string>
#include <vector>
#include <iostream>
 
//Executed on GPU

extern "C" {

    char* cudaHash(char* stringy) {
        JOB ** jobs;
        unsigned long temp;
        char * a_file = 0;
        BYTE * buff;
        int n = 0;

        a_file = stringy;

        buff = get_data(a_file, &temp);

        checkCudaErrors(hipMallocManaged(&jobs, 1 * sizeof(JOB *)));
        jobs[n++] = JOB_init(buff, temp, a_file);
    
        pre_sha256();
        runJobs(jobs, n);
        hipDeviceSynchronize();

        char * out;
        out = hash_to_string(jobs[0]->digest);

        // print_job(jobs[0]);

        return out;

        // hipDeviceReset();
        // int nDevices;
        // hipGetDeviceCount(&nDevices);
        // for (int i = 0; i < nDevices; i++) {
        //     hipDeviceProp_t prop;
        //     hipGetDeviceProperties(&prop, i);
        //     printf("Device Number: %d\n", i);
        //     printf("  Device name: %s\n", prop.name);
        //     printf("  Memory Clock Rate (KHz): %d\n",
        //         prop.memoryClockRate);
        //     printf("  Memory Bus Width (bits): %d\n",
        //         prop.memoryBusWidth);
        //     printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        //         2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // }
    }

//     

    void cudaMine(char* s, char* t, uint64_t nonce,
    char* d, char* u, int numMessages, uint64_t timestamp) {

    std::string source(s);
    std::string target(t);
    std::string data(d);
    std::string user(u);
    
    std::string datahash = cudaHash(d);
    ++nonce;
    int targetLength = target.length() / 2;
    if (target.length() % 2 != 0) {
        targetLength++;
    }

    BYTE * d_digest;
    BYTE * d_data;
    BYTE * d_data_init;
    BYTE * d_target;
    int * d_position;
    uint64_t * d_nonces;

    const int nonceSize = 20;

    std::vector<std::string> resultHashes(numMessages, "");
    std::string src;
    if (source.length() == 64 && source.find_first_not_of("0123456789abcdefABCDEF") == std::string::npos) {
        // source is a hash
        src = source;
    } else {
        src = cudaHash(s);
    }
    std::string message = src + datahash + target + user + std::to_string(timestamp);
    int messageSize = message.length() + nonceSize;
    int dataSize = sizeof(BYTE) * numMessages * messageSize;

    // Allocate device variables in Unified Memory
    checkCudaErrors(hipMallocManaged(&d_data_init, sizeof(BYTE) * message.length()));
    checkCudaErrors(hipMallocManaged(&d_target, sizeof(BYTE) * targetLength));
    checkCudaErrors(hipMallocManaged(&d_data, dataSize));
    checkCudaErrors(hipMallocManaged(&d_digest, sizeof(BYTE) * numMessages * 32));
    checkCudaErrors(hipMallocManaged(&d_position, sizeof(int) * numMessages));
    checkCudaErrors(hipMallocManaged(&d_nonces, sizeof(uint64_t) * numMessages));

    checkCudaErrors(hipMemcpy(d_data_init, &message.c_str()[0], sizeof(BYTE) * message.length(),
    hipMemcpyHostToDevice));

    // Convert target to hex values and add to d_target host variable
    for (int i = 0; i < target.length(); i += 2) {
        // if last two elements in string get last two values
        std::string str;
        str.append(1, target[i]);
        if (i + 1 < target.length()) str.append(1, target[i + 1]);
        d_target[i / 2] = std::stoi(str, 0, 16);
    }

    int blockSize = 512;
    bool * found;

    checkCudaErrors(hipMallocManaged(&found, sizeof(bool)));
    *found = false;

    int numBlocks = numMessages / blockSize;

    sha256_cuda<<<numBlocks, blockSize>>>(d_data, d_digest, numMessages, messageSize, d_target,
                targetLength, target.length(), d_position, d_data_init, nonce, found,
                d_nonces);
    checkCudaErrors(hipDeviceSynchronize());

    // Find value in resulthashes that has a length greater than 0 and return it
    for (int i = 0; i < numMessages; i++) {
        if (d_position[i] == 1) {
            BYTE bdata[32];
            int start = i * 32;
            int end = start + 32;
            // Convert byte hash to string
            for (int j = start; j < end; j++) {
                bdata[j % 32] = d_digest[j];
            }

            std::string foundhash = hash_to_string(bdata);
            std::cout << foundhash << std::endl;
            break;
        }
    }

    checkCudaErrors(hipFree(d_digest));
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_target));
    checkCudaErrors(hipFree(d_data_init));
    checkCudaErrors(hipFree(d_position));
}
}