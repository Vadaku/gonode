#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda/sha256.cuh"
#include "cuda/sha256.cu"
 
//Executed on GPU

extern "C" {

    char* getGPU(char* stringy) {
        JOB ** jobs;
        unsigned long temp;
        char * a_file = 0;
        BYTE * buff;
        int n = 0;

        a_file = stringy;

        buff = get_data(a_file, &temp);

        checkCudaErrors(hipMallocManaged(&jobs, 1 * sizeof(JOB *)));
        jobs[n++] = JOB_init(buff, temp, a_file);

        pre_sha256();
        runJobs(jobs, n);
        hipDeviceSynchronize();

        char * out;
        out = hash_to_string(jobs[0]->digest);

        // print_job(jobs[0]);

        return out;

        // hipDeviceReset();
        // int nDevices;
        // hipGetDeviceCount(&nDevices);
        // for (int i = 0; i < nDevices; i++) {
        //     hipDeviceProp_t prop;
        //     hipGetDeviceProperties(&prop, i);
        //     printf("Device Number: %d\n", i);
        //     printf("  Device name: %s\n", prop.name);
        //     printf("  Memory Clock Rate (KHz): %d\n",
        //         prop.memoryClockRate);
        //     printf("  Memory Bus Width (bits): %d\n",
        //         prop.memoryBusWidth);
        //     printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        //         2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        // }
    }
}