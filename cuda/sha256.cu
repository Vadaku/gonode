#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>

extern "C" void sha256_block_data_order (uint32_t *ctx, const void *in, size_t num);

char * trim(char *str){
    size_t len = 0;
    char *frontp = str;
    char *endp = NULL;

    if( str == NULL ) { return NULL; }
    if( str[0] == '\0' ) { return str; }

    len = strlen(str);
    endp = str + len;

    /* Move the front and back pointers to address the first non-whitespace
     * characters from each end.
     */
    while( isspace((unsigned char) *frontp) ) { ++frontp; }
    if( endp != frontp )
    {
        while( isspace((unsigned char) *(--endp)) && endp != frontp ) {}
    }

    if( str + len - 1 != endp )
            *(endp + 1) = '\0';
    else if( frontp != str &&  endp == frontp )
            *str = '\0';

    /* Shift the string so that it starts at str so that if it's dynamically
     * allocated, we can still free it on the returned pointer.  Note the reuse
     * of endp to mean the front of the string buffer now.
     */
    endp = str;
    if( frontp != str )
    {
            while( *frontp ) { *endp++ = *frontp++; }
            *endp = '\0';
    }


    return str;
}

__global__ void sha256_hash(JOB ** jobs, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	if (i < n){
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, jobs[i]->data, jobs[i]->size, i* jobs[i]->size);
		sha256_final(&ctx, jobs[i]->digest, i * 32);
	}
}

__global__ void sha256_cuda(BYTE * data, BYTE * digest, int n, int messageSize, BYTE * target,
        int targetHexCharCount, int targetLength, int * position, BYTE * d_data_init,
        uint64_t nonce, bool * found, uint64_t * nonces) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // perform sha256 calculation here
    if (i < n) {
        SHA256_CTX ctx;
        while (*found == false) {
            int buffer = i * messageSize;
            uint64_t tmp = nonce + i;

            // initialize nonce
            #pragma unroll
            for (int j = 1; j <= 20; j++) {
                data[buffer + messageSize - j] = (tmp % 10) + '0';
                tmp /= 10;
            }

            // initialize message data
            for (int j = 0; j < messageSize - 20; j++) {
                data[j + buffer] = d_data_init[j];
            }

            // hash data
            sha256_init(&ctx);
            sha256_update(&ctx, data, messageSize, i * messageSize);
            sha256_final(&ctx, digest, i * 32);

            position[i] = 1;

            // Check that the hash that is generated hash a valid target
            for (int j = 0; j < targetHexCharCount; j++) {
                unsigned int value = (unsigned int) digest[(i * 32) + j];
                // Check used if targetlength is an odd integer value
                if (targetLength % 2 != 0 && j == targetHexCharCount - 1) {
                    // Bitwise operation to check first value in hex val
                    if (value >> 4 != target[j]) {
                        position[i] = 0;
                        break;
                    }
                } else if (value != target[j]) {
                    // Check if hex values are not equal
                    position[i] = 0;
                    break;
                }
            }

            if (position[i] == 1) {
                nonces[i] = nonce + i;
                *found = true;
                return;
            }
            nonce += n;
        }
    }
}

void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

extern "C" {


void runJobs(JOB ** jobs, int n){
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;
	sha256_hash <<< numBlocks, blockSize >>> (jobs, n);
}

}


JOB * JOB_init(BYTE * data, long size, char * fname) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	strcpy(j->fname, fname);
	return j;
}


BYTE * get_file_data(char * fname, unsigned long * size) {
	FILE * f = 0;
	BYTE * buffer = 0;
	unsigned long fsize = 0;

	f = fopen(fname, "rb");
	if (!f){
		fprintf(stderr, "get_file_data Unable to open '%s'\n", fname);
		return 0;
	}
	fflush(f);

	if (fseek(f, 0, SEEK_END)){
		fprintf(stderr, "Unable to fseek %s\n", fname);
		return 0;
	}
	fflush(f);
	fsize = ftell(f);
	rewind(f);

	//buffer = (char *)malloc((fsize+1)*sizeof(char));
	checkCudaErrors(hipMallocManaged(&buffer, (fsize+1)*sizeof(char)));
	fread(buffer, fsize, 1, f);
	fclose(f);
	*size = fsize;
	return buffer;
}

BYTE * get_data(char* name, unsigned long * size){
	BYTE* buffer = 0;
	unsigned long ssize = 0;
	ssize = strlen(name);
	checkCudaErrors(hipMallocManaged(&buffer, (ssize+1)*sizeof(char)));
	memcpy(buffer, name, ssize+1);
	*size = ssize;
	return buffer;

}

void print_usage(){
	printf("Usage: CudaSHA256 [OPTION] [FILE]...\n");
	printf("Calculate sha256 hash of given FILEs\n\n");
	printf("OPTIONS:\n");
	printf("\t-f FILE1 \tRead a list of files (separeted by \\n) from FILE1, output hash for each file\n");
	printf("\t-h       \tPrint this help\n");
	printf("\nIf no OPTIONS are supplied, then program reads the content of FILEs and outputs hash for each FILEs \n");
	printf("\nOutput format:\n");
	printf("Hash following by two spaces following by file name (same as sha256sum).\n");
	printf("\nNotes:\n");
	printf("Calculations are performed on GPU, each seperate file is hashed in its own thread\n");
}